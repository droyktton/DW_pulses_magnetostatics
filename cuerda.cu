
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <tuple>


class ElasticLine {
public:
    double dt;
    double t;
    int L;
    std::vector<double> monomer;
    std::vector<double> fuerza;
    std::vector<double> disorder;

    double Temp = 0.0; // temperature
    double f0 = 0.6; // pulse amplitude
    double k0 = 0.0001; // magnetostatic constant
    double DisAmp = 1.0; // disorder amplitude
    double tau = 100.0; // pulse duration
    double tau0 = 10*tau; // ramp duration
    double f1 = 0.1; // ramp rate

    ElasticLine(int L_in) : L(L_in), dt(0.1), t(0.0) {
        monomer.resize(L, 0.0);
        fuerza.resize(L, 0.0);
        disorder.resize(L * L, 0.0);

        std::srand(0); // for reproducibility
        for (int i = 0; i < L; ++i) {
            for (int j = 0; j < L; ++j) {
                int n = i * L + j;
                disorder[n] = ((double)rand() / RAND_MAX * 2.0 - 1.0) * DisAmp;
            }
        }
    }

    // ramp and pulsated field
    double field() {
        int tmax = 3*tau0;
        if (t < tmax) {
            return f1 * t/ tmax; // ramp
        } else {
            return f0 * ((std::cos(t * 2*M_PI/tau) > 0) ? 1 : 0); // pulses
        }
    }

    void fuerzas() {
        for (int i = 0; i < L; ++i) {
            int ip1 = (i + 1) % L;
            int im1 = (i - 1 + L) % L;

            fuerza[i] = monomer[ip1] + monomer[im1] - 2 * monomer[i]
                        + field() - k0 * monomer[i];

            double u = monomer[i];
            u = u - std::floor(u / L) * L;
            int j = static_cast<int>(u) % L;

            int n = i * L + j;
            int np1 = i * L + (j + 1) % L;

            double interp_force = disorder[n] + (u - j) * (disorder[np1] - disorder[n]);
            fuerza[i] += interp_force;
        }
    }

    void update(int nrun) {
        for (int n = 0; n < nrun; ++n) {
            fuerzas();
            for (int i = 0; i < L; ++i) {
                monomer[i] += fuerza[i] * dt +
                              std::sqrt(Temp * dt) * (2.0 * rand() / RAND_MAX - 1.0);
            }
            t += dt;
        }
    }

    void reset() {
        std::fill(monomer.begin(), monomer.end(), 0.0);
        for (int i = 0; i < L; ++i) {
            for (int j = 0; j < L; ++j) {
                int n = i * L + j;
                disorder[n] = ((double)rand() / RAND_MAX * 2.0 - 1.0) * DisAmp;
            }
        }
        t = 0.0;
    }

    void print_config(std::ofstream &outputFile){
        for(int i = 0; i < L; ++i) {
            outputFile << monomer[i] << "\n";
        }
        outputFile << "\n" << std::endl;
    }

    void print_monitor(std::ofstream &monitorFile) {
      std::tuple<double, double> tup = cm_displacement();
      double cm = std::get<0>(tup);
      double var = std::get<1>(tup);
      monitorFile << t << " " << cm << " " << var << " " << f0 << "\n";
    }

    std::tuple<double, double> cm_displacement()
    {
        double cm = 0.0;
        for (int i = 0; i < L; ++i) cm += monomer[i];
        cm /= L;
        double var = 0.0;
        for (int i = 0; i < L; ++i) var += (monomer[i] - cm) * (monomer[i] - cm);
        var /= L;
        return std::tuple(cm, var);
    }
};

int main(int argc, char **argv) {

    int L = atoi(argv[1]);
    ElasticLine cuerda(L);

    std::ofstream outputFile("cuerda.dat");
    std::ofstream monitorFile("monitor.dat");
    std::ofstream monitorFile2("monitor2.dat");
    std::ofstream monitorstrobFile("monitorstrob.dat");

    // Simulation parameters
    cuerda.f0 = atof(argv[2]); // 0.4
    cuerda.k0 = atof(argv[3]); // 0.0001
    cuerda.DisAmp = atof(argv[4]); // 1.0
    cuerda.tau = atof(argv[5]); // 100.0
    cuerda.Temp = 0.0; // 0.0
    int nrun = 1;

    // Run simulation
    for (int step = 0; step < 200000; ++step) {

        double f0=cuerda.field();
        cuerda.update(nrun);
        double f1=cuerda.field();

	bool pulse = (f0<0.000001 && f1>0);

        //std::cout << f0 << " " << f1 << std::endl;
        if(pulse && step>0){
            //std::cout << "Step " << step << ", field = " << cuerda.field() << std::endl;
            cuerda.print_config(outputFile);
            cuerda.print_monitor(monitorstrobFile);
        }

        if(step%100==0)
        cuerda.print_monitor(monitorFile);

        if(pulse==1)
        cuerda.print_monitor(monitorFile2);
    }

    // Reset simulation
    // cuerda.reset();
}
