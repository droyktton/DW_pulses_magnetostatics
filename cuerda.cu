#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/fill.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <tuple>

__global__ void kernel_fuerzas(
    double* monomer,
    double* fuerza,
    const double* disorder,
    int L,
    double field,
    double k0)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= L) return;

    int ip1 = (i + 1) % L;
    int im1 = (i - 1 + L) % L;

    double f = monomer[ip1] + monomer[im1] - 2 * monomer[i]
               + field - k0 * monomer[i];

    double u = monomer[i];
    u = u - floor(u / L) * L;
    int j = static_cast<int>(u) % L;

    int n = i * L + j;
    int np1 = i * L + (j + 1) % L;

    double interp_force = disorder[n] + (u - j) * (disorder[np1] - disorder[n]);
    fuerza[i] = f + interp_force;
}

__global__ void kernel_update(
    double* monomer,
    const double* fuerza,
    double dt,
    double Temp,
    int L,
    hiprandState* states)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= L) return;

    double eta = hiprand_uniform_double(&states[i]) * 2.0 - 1.0;
    monomer[i] += fuerza[i] * dt + sqrt(Temp * dt) * eta;
}

__global__ void init_curand(hiprandState* states, unsigned long seed, int L)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < L)
        hiprand_init(seed, i, 0, &states[i]);
}

class ElasticLine {
public:
    double dt = 0.1;
    double t = 0.0;
    int L;

    thrust::device_vector<double> monomer;
    thrust::device_vector<double> fuerza;
    thrust::device_vector<double> disorder;

    double Temp = 0.0;
    double f0 = 0.6;
    double k0 = 0.0001;
    double DisAmp = 1.0;
    double tau = 100.0;
    double tau0;
    double f1 = 0.1;
    //bool pulses_on = false;
    double tmax = 3 * tau0;

    hiprandState* devStates;

    ElasticLine(int L_in) : L(L_in), tau0(10 * tau),
        monomer(L, 0.0),
        fuerza(L, 0.0),
        disorder(L * L)
    {
        // Initialize disorder on host
        thrust::host_vector<double> h_disorder(L * L);
        srand(0);
        for (int i = 0; i < L * L; ++i)
            h_disorder[i] = ((double)rand() / RAND_MAX * 2.0 - 1.0) * DisAmp;
        disorder = h_disorder;

        hipMalloc(&devStates, L * sizeof(hiprandState));
        init_curand<<<(L+255)/256, 256>>>(devStates, 1234, L);

        //pulses_on = false;
        tmax = 3 * tau0;

    }

    ~ElasticLine() {
        hipFree(devStates);
    }

    double field() {
        // tmax = 3 * tau0;
        // pulses_on = (t > tmax);
        if (t < tmax)
            return f1 * t / tmax;
        else{
            //pulses_on = true;
            return f0 * ((cos(t * 2 * M_PI / tau) > 0) ? 1 : 0);
        }
    }

    void fuerzas() {
        kernel_fuerzas<<<(L+255)/256, 256>>>(
            thrust::raw_pointer_cast(monomer.data()),
            thrust::raw_pointer_cast(fuerza.data()),
            thrust::raw_pointer_cast(disorder.data()),
            L, field(), k0
        );
    }

    void update(int nrun) {
        for (int n = 0; n < nrun; ++n) {
            fuerzas();
            kernel_update<<<(L+255)/256, 256>>>(
                thrust::raw_pointer_cast(monomer.data()),
                thrust::raw_pointer_cast(fuerza.data()),
                dt, Temp, L, devStates
            );
            t += dt;
        }
    }

    void print_config(std::ofstream &outputFile) {
        thrust::host_vector<double> h_monomer = monomer;
        for (int i = 0; i < L; ++i)
            outputFile << h_monomer[i] << "\n";
        outputFile << "\n" << std::endl;
    }

    std::tuple<double, double> cm_displacement() {
        thrust::host_vector<double> h_monomer = monomer;
        double cm = thrust::reduce(h_monomer.begin(), h_monomer.end()) / L;

        double var = thrust::transform_reduce(
            h_monomer.begin(), h_monomer.end(),
            [=] __host__ __device__ (double u) {
                return (u - cm) * (u - cm);
            },
            0.0, thrust::plus<double>()
        ) / L;

        return std::tuple(cm, var);
    }

    void print_monitor(std::ofstream &monitorFile) {
        auto [cm, var] = cm_displacement();
        monitorFile << t << " " << cm << " " << var << " " << f0 << "\n";
    }
};


int main(int argc, char **argv) {

    int L = atoi(argv[1]);
    ElasticLine cuerda(L);

    std::ofstream outputFile("cuerda.dat");
    std::ofstream monitorFile("monitor.dat");
    std::ofstream monitorFile2("monitor2.dat");
    std::ofstream monitorstrobFile("monitorstrob.dat");

    // Simulation parameters
    cuerda.f0 = atof(argv[2]); // 0.4
    cuerda.k0 = atof(argv[3]); // 0.0001
    cuerda.DisAmp = atof(argv[4]); // 1.0
    cuerda.tau = atof(argv[5]); // 100.0
    cuerda.Temp = atof(argv[6]); // 0.0
    int nrun = 1;

    std::cout << "L = " << L << std::endl;
    std::cout << "f0 = " << cuerda.f0 << std::endl;
    std::cout << "k0 = " << cuerda.k0 << std::endl;
    std::cout << "DisAmp = " << cuerda.DisAmp << std::endl;
    std::cout << "tau = " << cuerda.tau << std::endl;
    std::cout << "Temp = " << cuerda.Temp << std::endl;
    std::cout << "tmax = " << cuerda.tmax << std::endl;
    std::cout << std::endl;

    // Run simulation
    for (int step = 0; step < 200000; ++step) {

        double f0=cuerda.field();
        cuerda.update(nrun);
        double f1=cuerda.field();

	bool pulse = (f0<0.000001 && f1>0);

        //std::cout << f0 << " " << f1 << std::endl;
        if(pulse && step>0){
            //std::cout << "Step " << step << ", field = " << cuerda.field() << std::endl;
            cuerda.print_config(outputFile);
            cuerda.print_monitor(monitorstrobFile);
        }

        //if(step%100==0)
        //cuerda.print_monitor(monitorFile);

        if(pulse==true)
        cuerda.print_monitor(monitorFile2);
    }

    // Reset simulation
    // cuerda.reset();
}



